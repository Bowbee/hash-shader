#include "hip/hip_runtime.h"
#include "test.h"
#include <stdio.h>

#include "blocks_from_cli.h"

// linking cuda too hard
#include "sha256.cu"

void run_sha256(unsigned char *block_buf, int *block_starts, int num_blocks);

int main(int argc, char *argv[]) {
  if (argc < 2) {
	printf("Must have at least one argument\n");
	return -1;
  }
  struct Blocks b = blocks_from_cli(argc, argv);
  run_sha256((unsigned char *)b.block_buf, b.block_starts, b.num_blocks);
  free(b.block_starts);
  free(b.block_buf);
  return 0;
}

__global__ void kernel(unsigned char *block_buf, int *block_starts, int num_blocks, unsigned char* digests) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;;
  if (i < num_blocks) {
    SHA256(&block_buf[block_starts[i]], block_starts[i+1] - block_starts[i], &digests[i*SHA256_DIGEST_LENGTH]);
  }
}

void run_sha256(unsigned char *block_buf, int *block_starts, int num_blocks) {

  hipDeviceSynchronize();
  unsigned char *dev_block_buf;
  hipMallocManaged((void **)&dev_block_buf, block_starts[num_blocks]);
  hipMemcpy(dev_block_buf, block_buf, block_starts[num_blocks], hipMemcpyHostToDevice);

  int *dev_block_starts;
  hipMallocManaged((void **)&dev_block_starts, sizeof(int)*(num_blocks+1));
  hipMemcpy(dev_block_starts, block_starts, sizeof(int)*(num_blocks+1), hipMemcpyHostToDevice);

  unsigned char *dev_digests;
  hipMallocManaged((void **)&dev_digests, SHA256_DIGEST_LENGTH * num_blocks);
  unsigned char digests[SHA256_DIGEST_LENGTH * num_blocks] = {};

  kernel<<<1, num_blocks>>>(dev_block_buf, dev_block_starts, num_blocks, dev_digests);
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  hipMemcpy(digests, dev_digests, SHA256_DIGEST_LENGTH*num_blocks, hipMemcpyDeviceToHost);
  char res[num_blocks*SHA256_DIGEST_LENGTH + num_blocks] = "";
  int j = 0;
  for (int i = 0; i < num_blocks*SHA256_DIGEST_LENGTH; ++i) {
    if (i % SHA256_DIGEST_LENGTH == 0) {
      sprintf(&res[j], " ");
      j += 1;
    }
    sprintf(&res[j], "%02x", digests[i]);
    j += 2;
  }
  fprintf(stdout, "%s\n", res);

  hipFree(dev_block_buf);
  hipFree(dev_block_starts);
  hipFree(dev_digests);
}
