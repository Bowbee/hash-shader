#include "hip/hip_runtime.h"
#include "test.h"
#include <stdio.h>

// linking cuda too hard
#include "sha256.cu"
void run_sha256(unsigned char *d, int d_len, int N);

__global__ void kernel(unsigned char *d, int d_len, unsigned char *out) {
  SHA256(d, d_len, out);
}

int main(int argc, char *argv[]) {
  if (argc < 2) {
	printf("Must have at least one argument\n");
	return -1;
  }
  // TODO: work on several at strings at once
  run_sha256((unsigned char *)argv[1], strlen(argv[1]), (SHA256_DIGEST_LENGTH * 2));
  return 0;
}

void run_sha256(unsigned char *d, int d_len, int N) {
  hipDeviceSynchronize();
  unsigned char *d_c;
  hipMallocManaged((void **)&d_c, d_len);
  hipMemcpy(d_c, d, d_len, hipMemcpyHostToDevice);

  unsigned char *digest_c;
  hipMallocManaged((void **)&digest_c, SHA256_DIGEST_LENGTH);
  unsigned char digest[SHA256_DIGEST_LENGTH] = {};

  kernel<<<1, 1>>>(d_c, d_len, digest_c);
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  hipMemcpy(digest, digest_c, SHA256_DIGEST_LENGTH, hipMemcpyDeviceToHost);
  char res[N] = "";
  for (int i = 0; i < SHA256_DIGEST_LENGTH; ++i) {
    sprintf(res + i * 2, "%02x", digest_c[i]);
  }
  fprintf(stdout, "%s\n", res);

  hipFree(d_c);
  hipFree(digest_c);
}
