#include "hip/hip_runtime.h"
#include "test.h"
#include <stdio.h>

#include "blocks_from_csv.h"

// linking cuda too hard
#include "sha256.cu"

char* run_sha256(unsigned char *block_buf, int *block_starts, int num_blocks);

int main(int argc, char *argv[]) {
  if (argc < 2) {
    printf("Must have at least one argument\n");
    return -1;
  }
  struct Blocks b =  blocks_from_csv(argv[1]);

  /*
  printf("main:\n");
  for (int i = 0; i < b.block_starts[b.num_blocks]; i+=1) {
    int x = (unsigned char)b.block_buf[i];
    printf("%02x,", x);
  }
  printf("\n");
  for (int i = 0; i < b.num_blocks+1; ++i) {
    printf("%d ", b.block_starts[i]);
  }
  printf("\n");
  */

  char* _hashes = run_sha256((unsigned char *)b.block_buf, b.block_starts, b.num_blocks);
  free(b.block_starts);
  free(b.block_buf);

  strcmp(b.hashes[0], strtok(_hashes, " ")) != 0 ? printf("%d: True\n", 0) : printf("%d: False\n", 0);
  free(b.hashes[0]);
  for (int i = 1; i < b.num_blocks; ++i) {
    strcmp(b.hashes[i], strtok(NULL, " ")) != 0 ? printf("%d: True\n", i) : printf("%d: False\n", i);
    free(b.hashes[i]);
  }
  free(b.hashes);
  free(_hashes);
  return 0;
}

__global__ void kernel(unsigned char *block_buf, int *block_starts, int num_blocks, unsigned char* digests) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;;
  if (i < num_blocks) {
    SHA256(&block_buf[block_starts[i]], block_starts[i+1] - block_starts[i], &digests[i*SHA256_DIGEST_LENGTH]);
    SHA256(&digests[i*SHA256_DIGEST_LENGTH], SHA256_DIGEST_LENGTH, &digests[i*SHA256_DIGEST_LENGTH]);
  }
}

char* run_sha256(unsigned char *block_buf, int *block_starts, int num_blocks) {

  hipDeviceSynchronize();
  unsigned char *dev_block_buf;
  hipMallocManaged((void **)&dev_block_buf, block_starts[num_blocks]);
  hipMemcpy(dev_block_buf, block_buf, block_starts[num_blocks], hipMemcpyHostToDevice);

  int *dev_block_starts;
  hipMallocManaged((void **)&dev_block_starts, sizeof(int)*(num_blocks+1));
  hipMemcpy(dev_block_starts, block_starts, sizeof(int)*(num_blocks+1), hipMemcpyHostToDevice);

  unsigned char *dev_digests;
  hipMallocManaged((void **)&dev_digests, SHA256_DIGEST_LENGTH * num_blocks);
  unsigned char digests[SHA256_DIGEST_LENGTH * num_blocks] = {};

  kernel<<<1, num_blocks>>>(dev_block_buf, dev_block_starts, num_blocks, dev_digests);
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  hipMemcpy(digests, dev_digests, SHA256_DIGEST_LENGTH*num_blocks, hipMemcpyDeviceToHost);
  int res_len = (num_blocks * SHA256_DIGEST_LENGTH * 2) + num_blocks;
  char* res = (char*)malloc(res_len);
  int j = 0;
  for (int i = 0; i < num_blocks*SHA256_DIGEST_LENGTH; ++i) {
    if (i % SHA256_DIGEST_LENGTH == 0) {
      sprintf(&res[j], " ");
      j += 1;
    }
    sprintf(&res[j], "%02x", digests[i]);
    j += 2;
  }
  hipFree(dev_block_buf);
  hipFree(dev_block_starts);
  hipFree(dev_digests);
  return res;
}
