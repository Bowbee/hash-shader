#include "hip/hip_runtime.h"
#include "test.h"
#include <stdio.h>

#include "test_block_chain.h"

// linking cuda too hard
#include "sha256.cu"

char* run_sha256(unsigned char *block_buf, int *block_starts, int num_blocks);


void* pinned_alloc(size_t n) {
  void* h_aPinned = NULL;
  hipError_t status = hipHostMalloc((void**)&h_aPinned, n);
  if (status != hipSuccess) {
    printf("Error allocating pinned host memory\n");
    exit(-1);
  }
  return h_aPinned;
}

void pinned_free(void* p) {
  hipHostFree(p);
}

int main(int argc, char *argv[]) {
  if (argc < 2) {
    printf("Must have at least one argument\n");
    return -1;
  }

  test_block_chain(argv[1], argc > 2 ? atoi(argv[2]) : -1, run_sha256, pinned_alloc, pinned_free);
  return 0;
}

__global__ void kernel(unsigned char *block_buf, int *block_starts, int num_blocks, unsigned char* digests) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < num_blocks) {
    unsigned char intermidiate_digest[SHA256_DIGEST_LENGTH];
    int front = block_starts[i];
    int back = block_starts[i+1];
    SHA256(block_buf + front, back - front, intermidiate_digest);
    __syncthreads();
    SHA256(intermidiate_digest, SHA256_DIGEST_LENGTH, digests+(i*SHA256_DIGEST_LENGTH));
  }
}

char* run_sha256(unsigned char *block_buf, int *block_starts, int num_blocks) {

  hipDeviceSynchronize();
  unsigned char *dev_block_buf;
  hipMallocManaged((void **)&dev_block_buf, block_starts[num_blocks]);
  hipMemcpy(dev_block_buf, block_buf, block_starts[num_blocks], hipMemcpyHostToDevice);

  int *dev_block_starts;
  hipMallocManaged((void **)&dev_block_starts, sizeof(int)*(num_blocks+1));
  hipMemcpy(dev_block_starts, block_starts, sizeof(int)*(num_blocks+1), hipMemcpyHostToDevice);

  unsigned char *dev_digests;
  hipMallocManaged((void **)&dev_digests, SHA256_DIGEST_LENGTH * num_blocks);
  unsigned char digests[SHA256_DIGEST_LENGTH * num_blocks] = {};

  int num_thread_blocks = (num_blocks / 256) + 1;
  dim3 threadsPerThreadBlock(256);

  kernel<<<num_thread_blocks, threadsPerThreadBlock>>>(dev_block_buf, dev_block_starts, num_blocks, dev_digests);
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  hipMemcpy(digests, dev_digests, SHA256_DIGEST_LENGTH*num_blocks, hipMemcpyDeviceToHost);
  int res_len = (num_blocks * SHA256_DIGEST_LENGTH * 2) + num_blocks;
  char* res = (char*)malloc(res_len);
  int j = 0;
  for (int i = 0; i < num_blocks*SHA256_DIGEST_LENGTH; ++i) {
    if (i % SHA256_DIGEST_LENGTH == 0) {
      sprintf(&res[j], " ");
      j += 1;
    }
    sprintf(&res[j], "%02x", digests[i]);
    j += 2;
  }
  hipFree(dev_block_buf);
  hipFree(dev_block_starts);
  hipFree(dev_digests);
  return res;
}
