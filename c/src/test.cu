#include "hip/hip_runtime.h"
#include "test.h"
#include <stdio.h>

// linking cuda too hard
#include "sha256.cu"

static int test_idx = 0;
void test_sha256(unsigned char *d, int d_len, char *answer, int N);

__global__ void kernel(unsigned char *d, int d_len, unsigned char *out) {
  SHA256(d, d_len, out);
}

int main() {
  test(test_sha256, (SHA256_DIGEST_LENGTH * 2) + 1);
  return 0;
}

void test_sha256(unsigned char *d, int d_len, char *answer, int N) {
  hipDeviceSynchronize();
  unsigned char *d_c;
  hipMallocManaged((void **)&d_c, d_len);
  hipMemcpy(d_c, d, d_len, hipMemcpyHostToDevice);

  unsigned char *digest_c;
  hipMallocManaged((void **)&digest_c, SHA256_DIGEST_LENGTH);
  unsigned char digest[SHA256_DIGEST_LENGTH] = {};

  kernel<<<1, 1>>>(d_c, d_len, digest_c);
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  hipMemcpy(digest, digest_c, SHA256_DIGEST_LENGTH, hipMemcpyDeviceToHost);
  char res[N] = "";
  for (int i = 0; i < SHA256_DIGEST_LENGTH; ++i) {
    sprintf(res + i * 2, "%02x", digest_c[i]);
  }
  if (strcmp(res, answer) != 0) {
    fprintf(stderr,
            "cuda sha256 failed test %d\ngot:\n\t%s\n expected:\n\t%s\n\n",
            test_idx, res, answer);
  } else {
    fprintf(stdout,
            "cuda sha256 passed test %d\ngot:\n\t%s\n expected:\n\t%s\n\n",
            test_idx, res, answer);
  }
  test_idx += 1;
  hipFree(d_c);
  hipFree(digest_c);
}
